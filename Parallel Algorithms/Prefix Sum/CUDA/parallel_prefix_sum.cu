#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void parallelPrefixSum(int *input, int *output, int n) {
    extern __shared__ int temp[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        temp[threadIdx.x] = input[idx];
        __syncthreads();

        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int index = (threadIdx.x + 1) * 2 * stride - 1;
            if (index < blockDim.x) {
                temp[index] += temp[index - stride];
            }
            __syncthreads();
        }

        output[idx] = temp[threadIdx.x];
    }
}

int main() {
    int n = 8;
    int input[n] = {3, 1, 7, 0, 4, 1, 6, 3};
    int output[n];

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, n * sizeof(int));
    hipMalloc((void**)&d_output, n * sizeof(int));
    
    hipMemcpy(d_input, input, n * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 4;
    int grid_size = (n + block_size - 1) / block_size;

    parallelPrefixSum<<<grid_size, block_size, block_size * sizeof(int)>>>(d_input, d_output, n);

    hipMemcpy(output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d ", output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
